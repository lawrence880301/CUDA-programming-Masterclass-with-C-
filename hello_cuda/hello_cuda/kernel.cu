#include "hip/hip_runtime.h"


#include "stdio.h"

__global__ void hello_cuda() {
	printf("hello CUDA world \n");
}


int main() {
	//dim3 block(4);
	// means 4 threads per thread_block in x dimensiion

	//dim3 grid(8);
	//means 8 thread_blocks in x dimension

	int nx, ny;
	nx = 16;
	ny = 4;

	dim3 block(8, 2);
	dim3 grid(nx / block.x, ny / block.y);

	hello_cuda <<< grid,block >>> ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;

}